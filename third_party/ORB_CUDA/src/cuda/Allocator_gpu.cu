#include <cassert>
#include <hip/hip_runtime_api.h>
#include <cuda/Allocator.hpp>

namespace ORB_SLAM2 { namespace cuda {

size_t Allocator::getPitch(size_t widthSize){
    return 128 + widthSize - widthSize%128;
}

bool Allocator::allocate(cv::cuda::GpuMat* mat, int rows, int cols, size_t elemSize)
{
    if (rows > 1 && cols > 1)
    {
        mat->step = getPitch(elemSize * cols);
        checkCudaErrors(hipMallocManaged(&mat->data, mat->step * rows));
    }
    else
    {
        // Single row or single column must be continuous
        checkCudaErrors(hipMallocManaged(&mat->data, elemSize * cols * rows));
        mat->step = elemSize * cols;
    }

    mat->refcount = (int*) new int();

    return true;
}

void Allocator::free(cv::cuda::GpuMat* mat)
{
    checkCudaErrors(hipFree(mat->datastart));
    delete mat->refcount;
}

cv::cuda::GpuMat::Allocator * gpu_mat_allocator;

} }


namespace {
  using namespace ORB_SLAM2;

  void __attribute__((constructor)) init() {
    // Setup GPU Memory Management
    cuda::gpu_mat_allocator = new cuda::Allocator();
    // cv::cuda::GpuMat::setDefaultAllocator(cuda::gpu_mat_allocator);
  }
}
