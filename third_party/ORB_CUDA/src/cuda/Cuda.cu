#include <hip/hip_runtime_api.h>
#include <cuda/Cuda.hpp>

namespace ORB_SLAM2 { namespace cuda {
  void deviceSynchronize() {
    checkCudaErrors( hipDeviceSynchronize() );
  }
} }
